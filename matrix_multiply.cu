#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TILE_WIDTH 16

__global__ void matrixMulKernel(float* A, float* B, float* C, int numARows, int numAColumns, int numBColumns) {
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Cvalue = 0.0;

    for (int m = 0; m < (numAColumns + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        if (Row < numARows && m * TILE_WIDTH + tx < numAColumns) {
            ds_A[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];
        } else {
            ds_A[ty][tx] = 0.0;
        }

        if (Col < numBColumns && m * TILE_WIDTH + ty < numAColumns) {
            ds_B[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];
        } else {
            ds_B[ty][tx] = 0.0;
        }

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k) {
            Cvalue += ds_A[ty][k] * ds_B[k][tx];
        }

        __syncthreads();
    }

    if (Row < numARows && Col < numBColumns) {
        C[Row * numBColumns + Col] = Cvalue;
    }
}

void matrixMultiplyHost(float* A, float* B, float* C, int numARows, int numAColumns, int numBColumns) {
    for (int i = 0; i < numARows; ++i) {
        for (int j = 0; j < numBColumns; ++j) {
            float sum = 0.0;
            for (int k = 0; k < numAColumns; ++k) {
                sum += A[i * numAColumns + k] * B[k * numBColumns + j];
            }
            C[i * numBColumns + j] = sum;
        }
    }
}

void initializeMatrix(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        matrix[i] = (float)(rand() % 10);
    }
}

void printMatrix(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%.1f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}

int main(int argc, char *argv[]) {
    if (argc < 2) {
        printf("Usage: %s <dimenion of square matrix>\n", argv[0]);
        return 1;
    }


    int dimension = atoi(argv[1]);
    int numARows = dimension;
    int numAColumns = dimension;
    int numBRows = dimension;
    int numBColumns = dimension;

    size_t sizeA = numARows * numAColumns * sizeof(float);
    size_t sizeB = numBRows * numBColumns * sizeof(float);
    size_t sizeC = numARows * numBColumns * sizeof(float);

    float *h_A, *h_B, *h_C, *h_C_ref;
    float *d_A, *d_B, *d_C;

    h_A = (float*)malloc(sizeA);
    h_B = (float*)malloc(sizeB);
    h_C = (float*)malloc(sizeC);
    h_C_ref = (float*)malloc(sizeC);

    srand(time(NULL));
    initializeMatrix(h_A, numARows, numAColumns);
    initializeMatrix(h_B, numBRows, numBColumns);

    printf("Matrix A:\n");
    printMatrix(h_A, numARows, numAColumns);
    printf("\nMatrix B:\n");
    printMatrix(h_B, numBRows, numBColumns);

    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    dim3 dimGrid((numBColumns + TILE_WIDTH - 1) / TILE_WIDTH, (numARows + TILE_WIDTH - 1) / TILE_WIDTH);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matrixMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, numARows, numAColumns, numBColumns);
    hipEventRecord(stop);

    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float gpu_time;
    hipEventElapsedTime(&gpu_time, start, stop);

    clock_t cpu_start = clock();
    matrixMultiplyHost(h_A, h_B, h_C_ref, numARows, numAColumns, numBColumns);
    clock_t cpu_end = clock();
    float cpu_time = ((float)(cpu_end - cpu_start)) / CLOCKS_PER_SEC * 1000;

    printf("\nGPU Result (C = A * B):\n");
    printMatrix(h_C, numARows, numBColumns);

    printf("\nCPU Result (C = A * B):\n");
    printMatrix(h_C_ref, numARows, numBColumns);

    printf("\nMatrix dimensions: %dx%d * %dx%d = %dx%d\n", numARows, numAColumns, numBRows, numBColumns, numARows, numBColumns);
    printf("GPU time: %.2f ms\n", gpu_time);
    printf("CPU time: %.2f ms\n", cpu_time);
    printf("Speedup: %.2fx\n", cpu_time / gpu_time);

    bool correct = true;
    float tolerance = 1e-3;
    for (int i = 0; i < numARows * numBColumns && correct; ++i) {
        if (fabs(h_C[i] - h_C_ref[i]) > tolerance) {
            correct = false;
        }
    }

    if (correct) {
        printf("Results match!\n");
    } else {
        printf("Results do not match!\n");
    }

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_ref);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
